//
// Created by yuwang on 2020-04-22.
//

#include "matrix.h"
#include "cuda_operator.h"

#include <cstring>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

Matrix::Matrix() : _data(nullptr), _M(0), _N(0), _size(0) {};

Matrix::Matrix(int M, int N, bool zero) : _M(M), _N(N), _size(M * N) {
    assert(M > 0);
    assert(N > 0);

    // _data = new float[m._size];
    hipMallocManaged((void**)&_data, _size * sizeof(float));
    if (!zero) {
        for (int i = 0; i < _size; i++) {
            _data[i] = static_cast<float>(_rd());
        }
    }
}

Matrix::Matrix(float *data, int M, int N) : _data(data), _M(M), _N(N), _size(M * N) {
    assert(data != nullptr);
    assert(M > 0);
    assert(N > 0);
};

Matrix::Matrix(const Matrix &m) {
    _M = m._M;
    _N = m._N;
    _size = m._size;
    // _data = new float[m._size];
    hipMallocManaged((void**)&_data, _size * sizeof(float));
    memcpy(_data, m._data, m._size * sizeof(float));
}

Matrix::~Matrix() {
    if (_data != nullptr)
        hipFree(_data);
};

Matrix &Matrix::operator=(Matrix &&m) noexcept {
    if (&m == this) {
        return *this;
    }
    // this->_data = new float[m._size];
    // memcpy(this->_data, m._data, m._size * sizeof(float));
    this->_data = m._data;
    this->_M = m._M;
    this->_N = m._N;
    this->_size = m._size;
    
    m._data = nullptr;
    
    return *this;
}


int Matrix::size() {
    return _size;
}

int Matrix::size(int dim) {
    switch (dim) {
        case 0:
            return _M;
        case 1:
            return _N;
        default:
            throw dim;
    }
}

Matrix Matrix::T() {
    if (_size == 0) {
        return Matrix();
    }
    // auto *data = new float[this->_size];
    float *data;
    hipMallocManaged((void**)&data, _size * sizeof(float));
    auto M = _N;
    auto N = _M;

    for (int i = 0; i < _M; i++) {
        for (int j = 0; j < _N; j++) {
            data[j * _M + i] = this->_data[i * _N + j];
        }
    }

    return Matrix(data, M, N);
}

Matrix Matrix::add(Matrix &d) {
    assert(_M == d._M);
    assert(_N == d._N);

    // seq
    // auto *data = new float[_size];
    float *data;
    hipMallocManaged((void**)&data, _size * sizeof(float));
    // if (this->_dev == SEQ) {
    //     for (int i = 0; i < _M; i++) {
    //         for (int j = 0; j < _N; j++) {
    //             data[i * _N + j] = _data[i * _N + j] + d._data[i * _N + j];
    //         }
    //     }
    // }

    // cuda-parallel
    cuAdd(_data, d._data, data, _M, _N);

    return Matrix(data, _M, _N);
}

Matrix Matrix::dot(Matrix &d) {
    assert(_M == d._M);
    assert(_N == d._N);

    // auto *data = new float[_size];
    float *data;
    hipMallocManaged((void**)&data, _size * sizeof(float));
    // if (this->_dev == SEQ) {
    //     for (int i = 0; i < _M; i++) {
    //         for (int j = 0; j < _N; j++) {
    //             data[i * _N + j] = _data[i * _N + j] * d._data[i * _N + j];
    //         }
    //     }
    // }
    cuDot(_data, d._data, data, _M, _N);
    return Matrix(data, _M, _N);
}

Matrix Matrix::mul(Matrix &d) {
    /* assert dimension */
    assert(_N == d._M);

    /* do calculation */
    // auto *data = new float[_M * d._N];
    float *data;
    hipMallocManaged((void**)&data, _size * sizeof(float));
    // if (this->_dev == SEQ) {
    //     for (int i = 0; i < _M; i++) {
    //         for (int j = 0; j < d._N; j++) {
    //             for (int k = 0; k < _N; k++) {
    //                 data[i * d._N + j] +=
    //                         _data[i * _N + k] * d._data[k * d._N + j];
    //             }
    //         }
    //     }
    // }
    cuMul(_data, d._data, data, _M, d._N, _N);
    /* allocate new data */
    return Matrix(data, _M, d._N);
}

float *Matrix::data() {
    return _data;
}

Matrix Matrix::operator-() {
    float *data;
    hipMallocManaged((void**)&data, _size * sizeof(float));

    // for (int i = 0; i < _size; i++) {
    //     data[i] = -_data[i];
    // }
    cuNumMinus(_data, data, _size, 0);

    return Matrix(data, _M, _N);
}

Matrix Matrix::operator-(const float &num) {
    float *data;
    hipMallocManaged((void**)&data, _size * sizeof(float));

    // for (int i = 0; i < _size; i++) {
    //     data[i] = _data[i] - num;
    // }
    cuNumMinus(_data, data, _size, num);

    return Matrix(data, _M, _N);
}

Matrix Matrix::operator+(const float &num) {
    float *data;
    hipMallocManaged((void**)&data, _size * sizeof(float));

    // for (int i = 0; i < _size; i++) {
    //     data[i] = data[i] + num;
    // }
    cuNumAdd(_data, data, _size, num);

    return Matrix(data, _M, _N);
}

std::mt19937 Matrix::_rd = std::mt19937(0);