#include "hip/hip_runtime.h"
//
// Created by yuwang on 2020-04-21.
//

#include "tanh.h"
#include "cuda_operator.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

Matrix Tanh::forward(Matrix m) {
    // auto *__data = new float[m.size()];
    float *__data;
    hipMalloc((void**)&__data, m.size() * sizeof(float));

    float *_data = m.data();
    // if (m._dev == SEQ) {
    //     for (int i = 0; i < m.size(); i++) {
    //         __data[i] = tanh(_data[i]);
    //     }
    // }
    cuTanh(_data, __data, m.size());
    return std::move(Matrix(__data, m.size(0), m.size(1)));
}

