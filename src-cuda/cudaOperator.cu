#include "hip/hip_runtime.h"
#include <iostream>
#include "matrix.h"
#define INDEX(r, c, width) ((r) * (width) + (c))
#define MBLK 16
#define LBLK 32
#define MAXSIZE 1024
// cuda kernel variables
// store in fast memory
__constant__ float cuData[MAXSIZE];

static inline int updiv(int n, int d) {
    return (n + d - 1) / d;
}

__device__ static inline int sigmoid(double x) {
    return 1 / (1 + exp(x));
}

__device__ static inline double devTanh(double x) {
    return tanh(x);
}

// kernel functions
__global__ void cudaMatAddKernel(double* src1, double* src2, double* dst, int bound) {
    int i = blockIdx.x * blockDim.x+ threadIdx.x;
    if (i < bound){
        dst[i] = src1[i] + src2[i];
    }
}

__global__ void cudaMatDotKernel(double* src1, double* src2, double* dst, int bound) {
    int i = blockIdx.x * blockDim.x+ threadIdx.x;
    if (i < bound){
        dst[i] = src1[i] * src2[i];
    }
}

__global__ void cudaMatMulKernel(int M, int N, double* dmatA,double* dmatB, double * dmatC) {
    int i = blockIdx.x * blockDim.x+ threadIdx.x;
    int j = blockIdx.y * blockDim.y+ threadIdx.y;
    if (i>= M || j >= N) {
        return;
    }
    float sum = 0.0;
    for (int k = 0; k < N; k++) {
        sum += dmatA[INDEX(i,k,N)] * dmatB[INDEX(k,j,N)];
    }
    dmatC[INDEX(i,j,N)] = sum;
}

__global__ void cudaSigmoidKernel(double* src, double* dst, int length) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i > length) return;
    dst[i] = sigmoid(src[i]);
}

__global__ void cudaTanhKernel(double* src, double* dst, int length) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i > length) return;
    dst[i] = devTanh(src[i]);
}

// host functions
void CudaOperator::setup(int size, double* data) {
    std::cout<<"cuda setup...\n";
    hipMemcpy(cuData, data, sizeof(double) * size, hipMemcpyHostToDevice);
}

void CudaOperator::cuAdd(double* src1, double *src2, double* dst, int M, int N) {
    std::cout<<"cuAdd()\n";
    int elements = M * N;
    int size = elements * sizeof(double);
    // Allocate vectors in device memory
    double* d_A;
    hipMalloc(&d_A, size);
    double* d_B;
    hipMalloc(&d_B, size);
    double* d_C;
    hipMalloc(&d_C, size);
    
    // Copy vectors from host memory to device memory
    hipMemcpy(d_A, src1, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, src2, size, hipMemcpyHostToDevice);
    
    // Invoke kernel
    int threadsPerBlock = MBLK * MBLK;
    int blocksPerGrid = (elements + threadsPerBlock - 1) / threadsPerBlock;
    cudaMatAddKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, elements);
    
    // copy result
    hipMemcpy(dst, d_C, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

void CudaOperator::cuMul(double* A, double* B, double* C, int M, int N) {
    int elements = M * N;
    int size = elements * sizeof(double);
    // Allocate vectors in device memory
    double* d_A;
    hipMalloc(&d_A, size);
    double* d_B;
    hipMalloc(&d_B, size);
    double* d_C;
    hipMalloc(&d_C, size);

    // Copy vectors from host memory to device memory
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // Invoke Kernel
    dim3 threadsPerBlock(LBLK, LBLK);
    dim3 blocks(updiv(M, LBLK), updiv(N, LBLK));
    cudaMatMulKernel<<<blocks, threadsPerBlock>>>(M, N, d_A, d_B, d_C);

    // copy result
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

void CudaOperator::cuDot(double* A, double* B, double* C, int M, int N) {
    int elements = M * N;
    int threadsPerBlock = MBLK * MBLK;
    int blocksPerGrid = updiv(elements, threadsPerBlock);
    int size = elements * sizeof(double);
    // Allocate vectors in device memory
    double* d_A;
    hipMalloc(&d_A, size);
    double* d_B;
    hipMalloc(&d_B, size);
    double* d_C;
    hipMalloc(&d_C, size);

    // Copy matrix from host memory to device memory
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // Invoke
    cudaMatDotKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, elements);

    // copy result
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

void CudaOperator::cuSigmoid(double* src, double* dst, int length) {
    int size = length * sizeof(double);
    // Allocate vectors in device memory
    double* d_src;
    hipMalloc(&d_src, size);
    double* d_dst;
    hipMalloc(&d_dst, size);

    // Copy matrix from host memory to device memory
    hipMemcpy(d_src, src, size, hipMemcpyHostToDevice);

    // Invoke
    int threadsPerBlock = MBLK * MBLK;
    int blocksPerGrid = updiv(length, threadsPerBlock);
    cudaSigmoidKernel<<<blocksPerGrid, threadsPerBlock>>>(d_src, d_dst, length); 
    
    // copy result
    hipMemcpy(dst, d_dst, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_src);
    hipFree(d_dst);
}

void CudaOperator::cuTanh(double* src, double* dst, int length) {
    int size = length * sizeof(double);
    // Allocate vectors in device memory
    double* d_src;
    hipMalloc(&d_src, size);
    double* d_dst;
    hipMalloc(&d_dst, size);

    // Copy matrix from host memory to device memory
    hipMemcpy(d_src, src, size, hipMemcpyHostToDevice);

    int threadsPerBlock = MBLK * MBLK;
    int blocksPerGrid = updiv(length, threadsPerBlock);
    cudaSigmoidKernel<<<blocksPerGrid, threadsPerBlock>>>(d_src, d_dst, length);  
    
    // copy result
    hipMemcpy(dst, d_dst, size, hipMemcpyDeviceToHost); 

    // Free device memory
    hipFree(d_src);
    hipFree(d_dst);
}